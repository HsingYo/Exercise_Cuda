#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <math.h>
#include "kernel.h";

// Kernel function to add the elements of two arrays
__global__
void Std_Dev(int n, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = pow(x[i] - y[i], 2);

}

void cuda::FuncTest()
{
	int N = 1 << 25;
	float *x, *y;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = i % 256;
		y[i] = 255 - i % 256;
	}

	// Run kernel on 1M elements on the GPU
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	Std_Dev << <numBlocks, blockSize >> > (N, x, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	//Standard deviation
	float percent = 0.0f;
	for (int i = 0; i < N; i++) {
		percent += y[i];
	}
	percent = sqrt(percent / (N - 1));
	std::cout << "Percent: " << percent;
	// Free memory
	hipFree(x);
	hipFree(y);
}
